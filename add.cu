
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int a, int b, int *c){
	*c = a + b;
}

int main(void){
	int c;
	int *dev_c;
	hipMalloc((void**)&dev_c,sizeof(int));

	add<<<1,1>>> (6, 9,dev_c);

	hipMemcpy(&c,dev_c,sizeof(int),hipMemcpyDeviceToHost);
	printf("6+9=%d\n",c);
	hipFree(dev_c);
return 0;
}
